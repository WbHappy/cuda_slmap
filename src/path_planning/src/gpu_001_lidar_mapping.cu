#include "hip/hip_runtime.h"
#include "../include/gpu_001_lidar_mapping.cuh"

// GPU Kernel to used for LiDAR Mapping -> Creates raw heightmap of terrain
__global__ void lidarMappingKernel(
                            float* laser_scan,
                            const HTMatrixLidarCPU dk_cpu,
                            const int laser_rays,
                            const float angle_min,
                            const float angle_max,
                            int16_t* heightmap,
                            const int map_x,
                            const int map_y,
                            const int height_scale,
                            const int map_scale,
                            const float map_orient,
                            const float map_offset_pix,
                            float* debug)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // DISTANCE MEASURED BY LASER SCAN
    float a5 = laser_scan[tid];

    // ANGLE FROM MIDDLE OF SCANNING AREA
    float th5 = calcLaserAngle(laser_rays, angle_min, angle_max, tid);

    // GLOBAL POSITION OF POINT ON THE END OF THE SCAN
    Point3F32 point_world = dkLidarToScan(&dk_cpu, th5, a5);

    // POSITION OF SCAN POINT ON GPU HEIGHTMAP
    Point2I32 point_map = pointWorldToMap(point_world.x, point_world.y, map_orient, map_scale, map_offset_pix);

    // CHECKING IF SCAN POINT IS INSIDE GPU MAP
    if(point_map.x >=0 && point_map.x < map_x && point_map.y >=0 && point_map.y < map_y)
    {
        // ASSIGN NEW POINT TO GPU MAP
        heightmap[point_map.x * map_y + point_map.y] = (int16_t) (point_world.z * height_scale);
    }

}



// GPU function to calculate angle of scan depending on ID of GPU Thread
__device__ inline float calcLaserAngle(int laser_rays, float angle_min, float angle_max, int tid)
{
    return ((float)tid+0.5)/laser_rays*(angle_max - angle_min) + angle_min;
}

// GPU function to calculate direct kinematics from World to Scan,
// with given kinematics from World to Lidar
__device__ inline Point3F32 dkLidarToScan(const HTMatrixLidarCPU* dk_cpu, float th5, float a5)
{

    Point3F32 point;

    // A_GPU_14
    point.x = a5*dk_cpu->m_0*cos(th5) + a5*dk_cpu->m_1*sin(th5) + dk_cpu->m_3;

    // A_GPU_24
    point.y = a5*dk_cpu->m_4*cos(th5) + a5*dk_cpu->m_5*sin(th5) + dk_cpu->m_7;

    // A_GPU_34
    point.z = a5*dk_cpu->m_8*cos(th5) + a5*dk_cpu->m_9*sin(th5) + dk_cpu->m_11;

    return point;

}

// GPU function to transform given point from Real World into point on GPU Map
__device__ inline Point2I32 pointWorldToMap(float world_pose_x, float world_pose_y, float map_orient, float map_scale, float map_offset_pix)
{

    Point2I32 map_pose;
    float point_orient = atan2f(world_pose_y, world_pose_x);
    float point_dist = sqrtf(world_pose_x*world_pose_x + world_pose_y*world_pose_y);

    map_pose.x = (int) (sinf(map_orient - point_orient) * point_dist * map_scale + map_offset_pix);
    map_pose.y = (int) (cosf(map_orient - point_orient) * point_dist * map_scale + map_offset_pix);

    return map_pose;
}


GpuLidarMapping::GpuLidarMapping(_RobotPlannerMaps *_rpm, _ROSBuffor *_ros)
{
    this->_rpm = _rpm;
    this->_ros = _ros;
}



void GpuLidarMapping::allocateMemory(int laser_rays, float angle_min, float angle_max)
{
    this->laser_rays = laser_rays;
    this->angle_min = angle_min;
    this->angle_max = angle_max;

    gpuErrchk(hipMalloc((void**)&dev_laser_scan, laser_rays * sizeof(float)) );
    gpuErrchk(hipMalloc((void**)&dev_dk_matrix, 16 * sizeof(double)) );

}


void GpuLidarMapping::freeMemory()
{
    gpuErrchk( hipFree(dev_dk_matrix) );
    gpuErrchk( hipFree(dev_laser_scan) );

}


void GpuLidarMapping::drawInitialHeightmapCircle()
{
    _rpm->dev_heightmap.drawCircle(init_circle_height, _rpm->map_offset_pix, _rpm->map_offset_pix, init_circle_radius);
}



void GpuLidarMapping::copyInputToDevice()
{
    // Copying laser scan to GPU
    gpuErrchk( hipMemcpy(this->dev_laser_scan, &_ros->laser_scan.ranges[0], this->laser_rays * sizeof(float), hipMemcpyHostToDevice) );

}


void GpuLidarMapping::executeKernel()
{

    // CPU part
    this->dk_cpu = dkWorldToLidarReduced(
                    _ros->odom.pose.pose.position.x,
                    _ros->odom.pose.pose.position.y,
                    _ros->odom.pose.pose.position.z,
                    _ros->odom.pose.pose.orientation.x,
                    _ros->odom.pose.pose.orientation.y,
                    _ros->odom.pose.pose.orientation.z,
                    _ros->odom.pose.pose.orientation.w,
                    _ros->lidar_pose.data,
                    this->dk_a1,
                    this->dk_d2,
                    this->dk_al3);

    // GPU part
    lidarMappingKernel <<< this->laser_rays, 1 >>> (
                    this->dev_laser_scan,
                    this->dk_cpu,
                    this->laser_rays,
                    this->angle_min,
                    this->angle_max,
                    _rpm->dev_heightmap.data,
                    _rpm->dev_heightmap.size_x,
                    _rpm->dev_heightmap.size_y,
                    _rpm->height_scale,
                    _rpm->map_scale,
                    _rpm->map_orient,
                    _rpm->map_offset_pix,
                    _rpm->dev_debug);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}

void GpuLidarMapping::copyOutputToHost()
{
    gpuErrchk( hipMemcpy(_rpm->host_heightmap.data, _rpm->dev_heightmap.data, _rpm->dev_heightmap.size() * sizeof(int16_t), hipMemcpyDeviceToHost) );
}


void GpuLidarMapping::display()
{
    _rpm->host_heightmap.display("heightmap");
}


// CPU function to calculate direct kinematics form World to Lidar.
// Return full Homogenous Transformatin Matrix
HTMatrix dkWorldToLidar(
    double tx,
    double ty,
    double tz,
    double qx,
    double qy,
    double qz,
    double qw,
    double th2,
    const double a1,
    const double d2,
    const double al3
)
{

    OctaveVariable <double> TX(tx);
    OctaveVariable <double> TY(ty);
    OctaveVariable <double> TZ(tz);
    OctaveVariable <double> QX(qx);
    OctaveVariable <double> QY(qy);
    OctaveVariable <double> QZ(qz);
    OctaveVariable <double> QW(qw);
    OctaveVariable <double> TH2(th2);
    OctaveVariable <double> A1(a1);
    OctaveVariable <double> D2(d2);
    OctaveVariable <double> AL3(al3);

    OctaveVariable <double> TMP;

    HTMatrix dk_cpu;

    // A_CPU_11
    TMP =  -(2*QW*QY + 2*QX*QZ)*sin(AL3) + (-2*QW*QZ + 2*QX*QY)*(sin(TH2)*cos(AL3) + sin(TH2)/266709378811357127073829389436900 - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370) + (-2*QY->*2 - 2*QZ->*2 + 1)*(sin(TH2)*cos(AL3)/16331239353195370 - sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3) + cos(TH2)/266709378811357127073829389436900);
    dk_cpu.m[0][0] = TMP.data;


    // A_CPU_12
    TMP =  (2*QW*QY + 2*QX*QZ)*sin(AL3)/16331239353195370 + (-2*QW*QZ + 2*QX*QY)*(-sin(TH2)*cos(AL3)/16331239353195370 + sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3)/266709378811357127073829389436900 + cos(TH2)) + (-2*QY->*2 - 2*QZ->*2 + 1)*(-sin(TH2)*cos(AL3)/266709378811357127073829389436900 - sin(TH2) - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370);
    dk_cpu.m[0][1] = TMP.data;


    // A_CPU_13
    TMP =  (2*QW*QY + 2*QX*QZ)*cos(AL3) + (-2*QW*QZ + 2*QX*QY)*(sin(TH2)*sin(AL3) - sin(AL3)*cos(TH2)/16331239353195370) + (sin(TH2)*sin(AL3)/16331239353195370 + sin(AL3)*cos(TH2))*(-2*QY->*2 - 2*QZ->*2 + 1);
    dk_cpu.m[0][2] = TMP.data;


    // A_CPU_14
    TMP =  TX + A1*(-2*QY->*2 - 2*QZ->*2 + 1) + D2*(2*QW*QY + 2*QX*QZ);
    dk_cpu.m[0][3] = TMP.data;


    // A_CPU_21
    TMP =  -(-2*QW*QX + 2*QY*QZ)*sin(AL3) + (2*QW*QZ + 2*QX*QY)*(sin(TH2)*cos(AL3)/16331239353195370 - sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3) + cos(TH2)/266709378811357127073829389436900) + (-2*QX->*2 - 2*QZ->*2 + 1)*(sin(TH2)*cos(AL3) + sin(TH2)/266709378811357127073829389436900 - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370);
    dk_cpu.m[1][0] = TMP.data;


    // A_CPU_22
    TMP =  (-2*QW*QX + 2*QY*QZ)*sin(AL3)/16331239353195370 + (2*QW*QZ + 2*QX*QY)*(-sin(TH2)*cos(AL3)/266709378811357127073829389436900 - sin(TH2) - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370) + (-2*QX->*2 - 2*QZ->*2 + 1)*(-sin(TH2)*cos(AL3)/16331239353195370 + sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3)/266709378811357127073829389436900 + cos(TH2));
    dk_cpu.m[1][1] = TMP.data;


    // A_CPU_23
    TMP =  (-2*QW*QX + 2*QY*QZ)*cos(AL3) + (2*QW*QZ + 2*QX*QY)*(sin(TH2)*sin(AL3)/16331239353195370 + sin(AL3)*cos(TH2)) + (sin(TH2)*sin(AL3) - sin(AL3)*cos(TH2)/16331239353195370)*(-2*QX->*2 - 2*QZ->*2 + 1);
    dk_cpu.m[1][2] = TMP.data;


    // A_CPU_24
    TMP =  TY + A1*(2*QW*QZ + 2*QX*QY) + D2*(-2*QW*QX + 2*QY*QZ);
    dk_cpu.m[1][3] = TMP.data;


    // A_CPU_31
    TMP =  (2*QW*QX + 2*QY*QZ)*(sin(TH2)*cos(AL3) + sin(TH2)/266709378811357127073829389436900 - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370) + (-2*QW*QY + 2*QX*QZ)*(sin(TH2)*cos(AL3)/16331239353195370 - sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3) + cos(TH2)/266709378811357127073829389436900) - (-2*QX->*2 - 2*QY->*2 + 1)*sin(AL3);
    dk_cpu.m[2][0] = TMP.data;


    // A_CPU_32
    TMP =  (2*QW*QX + 2*QY*QZ)*(-sin(TH2)*cos(AL3)/16331239353195370 + sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3)/266709378811357127073829389436900 + cos(TH2)) + (-2*QW*QY + 2*QX*QZ)*(-sin(TH2)*cos(AL3)/266709378811357127073829389436900 - sin(TH2) - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370) + (-2*QX->*2 - 2*QY->*2 + 1)*sin(AL3)/16331239353195370;
    dk_cpu.m[2][1] = TMP.data;


    // A_CPU_33
    TMP =  (2*QW*QX + 2*QY*QZ)*(sin(TH2)*sin(AL3) - sin(AL3)*cos(TH2)/16331239353195370) + (-2*QW*QY + 2*QX*QZ)*(sin(TH2)*sin(AL3)/16331239353195370 + sin(AL3)*cos(TH2)) + (-2*QX->*2 - 2*QY->*2 + 1)*cos(AL3);
    dk_cpu.m[2][2] = TMP.data;


    // A_CPU_34
    TMP =  TZ + A1*(-2*QW*QY + 2*QX*QZ) + D2*(-2*QX->*2 - 2*QY->*2 + 1);
    dk_cpu.m[2][3] = TMP.data;


    // A_CPU_41
    dk_cpu.m[3][0] =  0;


    // A_CPU_42
    dk_cpu.m[3][1] =  0;


    // A_CPU_43
    dk_cpu.m[3][2] =  0;


    // A_CPU_44
    dk_cpu.m[3][3] =  1;


    return dk_cpu;

}


// CPU function to calculate direct kinematics form World to Lidar.
// Return reduced Homogenous Transformatin Matrix (9 instead of 16 elements)
HTMatrixLidarCPU dkWorldToLidarReduced(
    double tx,
    double ty,
    double tz,
    double qx,
    double qy,
    double qz,
    double qw,
    double th2,
    const double a1,
    const double d2,
    const double al3
)
{

        OctaveVariable <double> TX(tx);
        OctaveVariable <double> TY(ty);
        OctaveVariable <double> TZ(tz);
        OctaveVariable <double> QX(qx);
        OctaveVariable <double> QY(qy);
        OctaveVariable <double> QZ(qz);
        OctaveVariable <double> QW(qw);
        OctaveVariable <double> TH2(th2);
        OctaveVariable <double> A1(a1);
        OctaveVariable <double> D2(d2);
        OctaveVariable <double> AL3(al3);

        OctaveVariable <double> TMP;

        HTMatrixLidarCPU dk_cpu;

        // A_CPU_11
        TMP =  -(2*QW*QY + 2*QX*QZ)*sin(AL3) + (-2*QW*QZ + 2*QX*QY)*(sin(TH2)*cos(AL3) + sin(TH2)/266709378811357127073829389436900 - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370) + (-2*QY->*2 - 2*QZ->*2 + 1)*(sin(TH2)*cos(AL3)/16331239353195370 - sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3) + cos(TH2)/266709378811357127073829389436900);
        dk_cpu.m_0 = TMP.data;


        // A_CPU_12
        TMP =  (2*QW*QY + 2*QX*QZ)*sin(AL3)/16331239353195370 + (-2*QW*QZ + 2*QX*QY)*(-sin(TH2)*cos(AL3)/16331239353195370 + sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3)/266709378811357127073829389436900 + cos(TH2)) + (-2*QY->*2 - 2*QZ->*2 + 1)*(-sin(TH2)*cos(AL3)/266709378811357127073829389436900 - sin(TH2) - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370);
        dk_cpu.m_1 = TMP.data;


        // A_CPU_14
        TMP =  TX + A1*(-2*QY->*2 - 2*QZ->*2 + 1) + D2*(2*QW*QY + 2*QX*QZ);
        dk_cpu.m_3 = TMP.data;


        // A_CPU_21
        TMP =  -(-2*QW*QX + 2*QY*QZ)*sin(AL3) + (2*QW*QZ + 2*QX*QY)*(sin(TH2)*cos(AL3)/16331239353195370 - sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3) + cos(TH2)/266709378811357127073829389436900) + (-2*QX->*2 - 2*QZ->*2 + 1)*(sin(TH2)*cos(AL3) + sin(TH2)/266709378811357127073829389436900 - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370);
        dk_cpu.m_4 = TMP.data;


        // A_CPU_22
        TMP =  (-2*QW*QX + 2*QY*QZ)*sin(AL3)/16331239353195370 + (2*QW*QZ + 2*QX*QY)*(-sin(TH2)*cos(AL3)/266709378811357127073829389436900 - sin(TH2) - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370) + (-2*QX->*2 - 2*QZ->*2 + 1)*(-sin(TH2)*cos(AL3)/16331239353195370 + sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3)/266709378811357127073829389436900 + cos(TH2));
        dk_cpu.m_5 = TMP.data;


        // A_CPU_24
        TMP =  TY + A1*(2*QW*QZ + 2*QX*QY) + D2*(-2*QW*QX + 2*QY*QZ);
        dk_cpu.m_7 = TMP.data;


        // A_CPU_31
        TMP =  (2*QW*QX + 2*QY*QZ)*(sin(TH2)*cos(AL3) + sin(TH2)/266709378811357127073829389436900 - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370) + (-2*QW*QY + 2*QX*QZ)*(sin(TH2)*cos(AL3)/16331239353195370 - sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3) + cos(TH2)/266709378811357127073829389436900) - (-2*QX->*2 - 2*QY->*2 + 1)*sin(AL3);
        dk_cpu.m_8 = TMP.data;


        // A_CPU_32
        TMP =  (2*QW*QX + 2*QY*QZ)*(-sin(TH2)*cos(AL3)/16331239353195370 + sin(TH2)/16331239353195370 + cos(TH2)*cos(AL3)/266709378811357127073829389436900 + cos(TH2)) + (-2*QW*QY + 2*QX*QZ)*(-sin(TH2)*cos(AL3)/266709378811357127073829389436900 - sin(TH2) - cos(TH2)*cos(AL3)/16331239353195370 + cos(TH2)/16331239353195370) + (-2*QX->*2 - 2*QY->*2 + 1)*sin(AL3)/16331239353195370;
        dk_cpu.m_9 = TMP.data;


        // A_CPU_34
        TMP =  TZ + A1*(-2*QW*QY + 2*QX*QZ) + D2*(-2*QX->*2 - 2*QY->*2 + 1);
        dk_cpu.m_11 = TMP.data;

        return dk_cpu;
}
