#include "hip/hip_runtime.h"
#include "../include/gpu_map_ui32.cuh"

__global__ void fillValueKernel(uint32_t* data, const uint32_t fill_value)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = idy + idx * gridDim.y * blockDim.y;

    data[tid] = fill_value;
}

__global__ void drawCircleKernel(uint32_t* data, const uint32_t fill_value, const int pose_x, const int pose_y, const float radius)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = idy + idx * gridDim.y * blockDim.y;

    int dist_x = (idx - pose_x);
    int dist_y = (idy - pose_y);

    float dist_from_center = sqrtf((float)dist_x*dist_x + (float)dist_y*dist_y);

    if(dist_from_center <= radius)
    {
        data[tid] = fill_value;
    }

}

__global__ void drawBordersKernel(uint32_t* data, const uint32_t fill_value, const int thickness)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = idy + idx * gridDim.y * blockDim.y;

    if(
        idx < thickness ||
        idx > gridDim.x * blockDim.x - thickness ||
        idy < thickness ||
        idy > gridDim.y * blockDim.y - thickness
    )
    {
        data[tid] = fill_value;
    }

}

GpuMapUI32::GpuMapUI32()
{
    size_x = 0;
    size_y = 0;
}

GpuMapUI32::GpuMapUI32(int size_x, int size_y)
{
    this->resize(size_x, size_y);
}


GpuMapUI32::GpuMapUI32(int size_x, int size_y, const uint32_t fill_value)
{
    this->resize(size_x, size_y);
    this->fill(fill_value);
}


void GpuMapUI32::allocate(int size_x, int size_y)
{
    this->size_x = size_x;
    this->size_y = size_y;

    gpuErrchk( hipMalloc((void**)&this->data, size_x * size_y * sizeof(uint32_t)) );
}


void GpuMapUI32::resize(int size_x, int size_y)
{
    release();
    allocate(size_x, size_y);
}


void GpuMapUI32::fill(const uint32_t fill_value)
{
    int block_x = 32;
    int block_y = 32;

    int grid_x = (size_x + block_x - 1) / block_x;
    int grid_y = (size_y + block_y - 1) / block_y;
    dim3 grid(grid_x, grid_y, 1);
    dim3 block(block_x, block_y, 1);

    fillValueKernel<<< grid, block >>> (this->data, fill_value);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}

void GpuMapUI32::drawCircle(const uint32_t fill_value, const int pose_x, const int pose_y, const float radius)
{
    int block_x = 32;
    int block_y = 32;

    int grid_x = (size_x + block_x - 1) / block_x;
    int grid_y = (size_y + block_y - 1) / block_y;
    dim3 grid(grid_x, grid_y, 1);
    dim3 block(block_x, block_y, 1);

    drawCircleKernel<<< grid, block >>> (this->data, fill_value, pose_x, pose_y, radius);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}


void GpuMapUI32::drawBorders(const uint32_t fill_value, const int thickness)
{
    int block_x = 32;
    int block_y = 32;

    int grid_x = (size_x + block_x - 1) / block_x;
    int grid_y = (size_y + block_y - 1) / block_y;
    dim3 grid(grid_x, grid_y, 1);
    dim3 block(block_x, block_y, 1);

    drawBordersKernel<<< grid, block >>> (this->data, fill_value, thickness);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}

void GpuMapUI32::release()
{
    gpuErrchk( hipFree(this->data) );
}
